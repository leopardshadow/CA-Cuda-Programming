#include "hip/hip_runtime.h"
#include "parameters.h"


__global__ void cuda_kernel(int *B,int *A,IndexSave *dInd)
{	
	//
};


float GPU_kernel(int *B,int *A,IndexSave* indsave){

	int *dA,*dB;
	IndexSave* dInd;

	// Creat Timing Event
  	hipEvent_t start, stop;
	hipEventCreate (&start);
	hipEventCreate (&stop); 	

	// Allocate Memory Space on Device
	hipMalloc((void**)&dA,sizeof(int)*SIZE);
	hipMalloc((void**)&dB,sizeof(int)*SIZE);

	// Allocate Memory Space on Device (for observation)
	hipMalloc((void**)&dInd,sizeof(IndexSave)*SIZE);

	// Copy Data to be Calculated
	hipMemcpy(dA, A, sizeof(int)*SIZE, hipMemcpyHostToDevice);
	hipMemcpy(dB, A, sizeof(int)*SIZE, hipMemcpyHostToDevice);

	// Copy Data (indsave array) to device
	hipMemcpy(dInd, indsave, sizeof(IndexSave)*SIZE, hipMemcpyHostToDevice);
	
	// Start Timer
	hipEventRecord(start, 0);

	// Lunch Kernel
	dim3 dimGrid(2);
	dim3 dimBlock(4);
	cuda_kernel<<<dimGrid,dimBlock>>>(dB,dA,dInd);

	// Stop Timer
	hipEventRecord(stop, 0);
  	hipEventSynchronize(stop); 

	// Copy Output back
	hipMemcpy(indsave, dInd, sizeof(IndexSave)*SIZE, hipMemcpyDeviceToHost);
	hipMemcpy(B, dB, sizeof(int)*SIZE, hipMemcpyDeviceToHost);

	// Release Memory Space on Device
	hipFree(dA);
	hipFree(dB);
	hipFree(dInd);

	// Calculate Elapsed Time
  	float elapsedTime; 
  	hipEventElapsedTime(&elapsedTime, start, stop);  

	return elapsedTime;
}
