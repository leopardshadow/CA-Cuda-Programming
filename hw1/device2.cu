#include "hip/hip_runtime.h"
#include "parameters.h"


__global__ void cuda_kernel(int *B,int *A,IndexSave *dInd)
{	
	//
	int i = 0;
	int totalThread = blockDim.x * gridDim.x;
	int stripe = totalThread;
	int head = blockDim.x * blockIdx.x + threadIdx.x;
	for(i = head; i < SIZE; i+=stripe) {
		dInd[i].blockInd_x = blockIdx.x;
		dInd[i].threadInd_x = threadIdx.x;
		dInd[i].head = head;
		dInd[i].stripe = stripe;
		for(int j=1;j<LOOP;j++) {
			B[i]*=A[i];
		}
	}
};


float GPU_kernel(int *B,int *A,IndexSave* indsave){

	int *dA,*dB;
	IndexSave* dInd;

	// Creat Timing Event
  	hipEvent_t start, stop;
	hipEventCreate (&start);
	hipEventCreate (&stop); 	

	// Allocate Memory Space on Device
	hipMalloc((void**)&dA,sizeof(int)*SIZE);
	hipMalloc((void**)&dB,sizeof(int)*SIZE);

	// Allocate Memory Space on Device (for observation)
	hipMalloc((void**)&dInd,sizeof(IndexSave)*SIZE);

	// Copy Data to be Calculated
	hipMemcpy(dA, A, sizeof(int)*SIZE, hipMemcpyHostToDevice);
	hipMemcpy(dB, A, sizeof(int)*SIZE, hipMemcpyHostToDevice);

	// Copy Data (indsave array) to device
	hipMemcpy(dInd, indsave, sizeof(IndexSave)*SIZE, hipMemcpyHostToDevice);
	
	// Start Timer
	hipEventRecord(start, 0);

	// Lunch Kernel
	dim3 dimGrid(2);
	dim3 dimBlock(4);
	cuda_kernel<<<dimGrid,dimBlock>>>(dB,dA,dInd);

	// Stop Timer
	hipEventRecord(stop, 0);
  	hipEventSynchronize(stop); 

	// Copy Output back
	hipMemcpy(indsave, dInd, sizeof(IndexSave)*SIZE, hipMemcpyDeviceToHost);
	hipMemcpy(B, dB, sizeof(int)*SIZE, hipMemcpyDeviceToHost);

	// Release Memory Space on Device
	hipFree(dA);
	hipFree(dB);
	hipFree(dInd);

	// Calculate Elapsed Time
  	float elapsedTime; 
  	hipEventElapsedTime(&elapsedTime, start, stop);  

	return elapsedTime;
}
