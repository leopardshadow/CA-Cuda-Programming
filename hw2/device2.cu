#include "hip/hip_runtime.h"
#include "parameters.h"
#include <stdio.h>

__global__ void gpu_simulate(u_int8_t *dNow, u_int8_t *dNext)
{	
    int i, j;
    int t, adjac;

    int stripeI = N / blockNum;
    int stripeJ = N / gridNum;

    // printf("Stripe: i = %d, j = %d\n", stripeI, stripeJ);

    for(i = threadIdx.x + 1; i <= N; i+=stripeI) {
        for(j = blockIdx.x + 1; j <= N; j+=stripeJ) {

            printf("i = %d, j = %d\n", i, j);

            adjac = 
                dNow[T(i-1, j-1)] + dNow[T(i, j-1)] + dNow[T(i+1, j-1)] + 
                dNow[T(i-1, j)]   +                 + dNow[T(i+1, j)]   + 
                dNow[T(i-1, j+1)] + dNow[T(i, j+1)] + dNow[T(i+1, j+1)];
    
    
            t = dNow[T(i, j)];
        
            if((!t && adjac == 3) || (t && adjac == 2) || (t && adjac == 3)) {
                dNext[T(i, j)] = 1;
            }
            else {
                dNext[T(i, j)] = 0;
            }
        
        }
    }
    
}

u_int8_t *runGPUSimulations(u_int8_t event[2][(N+2)*(N+2)]) {
    
    u_int8_t *gpuEvent[2];

	hipMalloc((void**)&gpuEvent[0], sizeof(u_int8_t)*((N+2)*(N+2)));
	hipMalloc((void**)&gpuEvent[1], sizeof(u_int8_t)*((N+2)*(N+2)));

    hipMemcpy(gpuEvent[0], event[0], sizeof(u_int8_t)*((N+2)*(N+2)), hipMemcpyHostToDevice);
    hipMemcpy(gpuEvent[1], event[1], sizeof(u_int8_t)*((N+2)*(N+2)), hipMemcpyHostToDevice);

    // Launch Kernel
	dim3 dimGrid(gridNum);
    dim3 dimBlock(blockNum);

    for(int m = 0; m < M; m++) {
        gpu_simulate<<<dimGrid,dimBlock>>>(gpuEvent[m%2], gpuEvent[(m+1)%2]);
    }

	hipMemcpy(event[0], gpuEvent[0], sizeof(u_int8_t)*((N+2)*(N+2)), hipMemcpyDeviceToHost);
	hipMemcpy(event[1], gpuEvent[1], sizeof(u_int8_t)*((N+2)*(N+2)), hipMemcpyDeviceToHost);

    return event[M%2];
}
