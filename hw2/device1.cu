#include "hip/hip_runtime.h"
#include "parameters.h"

__global__ void gpu_simulate(u_int8_t *dNow, u_int8_t *dNext)
{	
    int i, j;
    int t, adjac;

    int limI = N / blockNum;
    int limJ = N / gridNum;

    for(i = threadIdx.x; i <= threadIdx.x + limI; i++) {
        for(j = blockIdx.x; j <= blockIdx.x + limJ; j++) {

            adjac = 
                dNow[T(i-1, j-1)] + dNow[T(i, j-1)] + dNow[T(i+1, j-1)] + 
                dNow[T(i-1, j)]   +                 + dNow[T(i+1, j)]   + 
                dNow[T(i-1, j+1)] + dNow[T(i, j+1)] + dNow[T(i+1, j+1)];
    
    
            t = dNow[T(i, j)];
        
            if((!t && adjac == 3) || (t && adjac == 2) || (t && adjac == 3)) {
                dNext[T(i, j)] = 1;
            }
            else {
                dNext[T(i, j)] = 0;
            }
        
        }
    }
    
}

u_int8_t *runGPUSimulations(u_int8_t event[2][(N+2)*(N+2)]) {
    
    u_int8_t *gpuEvent[2];

	hipMalloc((void**)&gpuEvent[0], sizeof(u_int8_t)*((N+2)*(N+2)));
	hipMalloc((void**)&gpuEvent[1], sizeof(u_int8_t)*((N+2)*(N+2)));

    hipMemcpy(gpuEvent[0], event[0], sizeof(u_int8_t)*((N+2)*(N+2)), hipMemcpyHostToDevice);
    hipMemcpy(gpuEvent[1], event[1], sizeof(u_int8_t)*((N+2)*(N+2)), hipMemcpyHostToDevice);

    // Launch Kernel
	dim3 dimGrid(gridNum);
    dim3 dimBlock(blockNum);

    for(int m = 0; m < M; m++) {
        gpu_simulate<<<dimGrid,dimBlock>>>(gpuEvent[m%2], gpuEvent[(m+1)%2]);
    }

	hipMemcpy(event[0], gpuEvent[0], sizeof(u_int8_t)*((N+2)*(N+2)), hipMemcpyDeviceToHost);
	hipMemcpy(event[1], gpuEvent[1], sizeof(u_int8_t)*((N+2)*(N+2)), hipMemcpyDeviceToHost);

    return event[M%2];
}
