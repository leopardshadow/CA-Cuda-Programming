#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

#include "parameters.h"

#define T(x, y) ((x) + (N+2)*(y))

extern float GPU_kernel(int *B, int *A);

u_int8_t state[N][N]; 
// = {
//     {'1','0','0','0','1'},
//     {'0','0','1','0','0'},
//     {'0','1','1','1','0'},
//     {'0','0','1','0','0'},
//     {'0','1','0','1','0'},
// };

u_int8_t event[2][ (N+2) * (N+2) ] =  {0};

u_int8_t *gpuEvent[2];

u_int8_t *result;

void simulate(u_int8_t *now, u_int8_t *next) {
    int i, j, t;
    int adjac;
    for(i = 1; i <= N; i++) {
        for(j = 1; j <= N; j++) {

            adjac = 
                now[T(i-1, j-1)] + now[T(i, j-1)] + now[T(i+1, j-1)] + 
                now[T(i-1, j)]   +                + now[T(i+1, j)]   + 
                now[T(i-1, j+1)] + now[T(i, j+1)] + now[T(i+1, j+1)];


            t = now[T(i, j)];

            if((!t && adjac == 3) || (t && adjac == 2) || (t && adjac == 3)) {
                next[T(i, j)] = 1;
            }
            else {
                next[T(i, j)] = 0;
            }
            // printf("%d %d: %c - %d\n", i, j, '0'+next[T(i, j)], adjac);
        }

    }
}


void runSimulations() {
    
    // do simulation m times
    for(int m = 0; m < M; m++) {

        simulate(event[m%2], event[(m+1)%2]);
    }
    result = event[M%2];
}

__global__ void gpu_simulate(u_int8_t *dNow, u_int8_t *dNext)
{	
    int i, j;
    int t, adjac;

    int stripeI = N / blockNum;
    int stripeJ = N / gridNum;

    // printf("Stripe: i = %d, j = %d\n", stripeI, stripeJ);

    for(i = threadIdx.x; i <= N; i+=stripeI) {
        for(j = blockIdx.x; j <= N; j+=stripeJ) {

            // printf("i = %d, j = %d\n", i, j);

            adjac = 
                dNow[T(i-1, j-1)] + dNow[T(i, j-1)] + dNow[T(i+1, j-1)] + 
                dNow[T(i-1, j)]   +                 + dNow[T(i+1, j)]   + 
                dNow[T(i-1, j+1)] + dNow[T(i, j+1)] + dNow[T(i+1, j+1)];
    
    
            t = dNow[T(i, j)];
        
            if((!t && adjac == 3) || (t && adjac == 2) || (t && adjac == 3)) {
                dNext[T(i, j)] = 1;
            }
            else {
                dNext[T(i, j)] = 0;
            }
        
        }
    }
    
}



void runGPUSimulations() {
    

	hipMalloc((void**)&gpuEvent[0], sizeof(u_int8_t)*((N+2)*(N+2)));
	hipMalloc((void**)&gpuEvent[1], sizeof(u_int8_t)*((N+2)*(N+2)));

    hipMemcpy(gpuEvent[0], event[0], sizeof(u_int8_t)*((N+2)*(N+2)), hipMemcpyHostToDevice);
    hipMemcpy(gpuEvent[1], event[1], sizeof(u_int8_t)*((N+2)*(N+2)), hipMemcpyHostToDevice);

    // Launch Kernel
	dim3 dimGrid(gridNum);
    dim3 dimBlock(blockNum);

    for(int m = 0; m < M; m++) {
        gpu_simulate<<<dimGrid,dimBlock>>>(gpuEvent[m%2], gpuEvent[(m+1)%2]);
    }

	hipMemcpy(event[0], gpuEvent[0], sizeof(u_int8_t)*((N+2)*(N+2)), hipMemcpyDeviceToHost);
	hipMemcpy(event[1], gpuEvent[1], sizeof(u_int8_t)*((N+2)*(N+2)), hipMemcpyDeviceToHost);

    result = event[M%2];
}



int main(int argc, char const *argv[])
{
    int i, j;

    state[5][3] = 1;
    state[5][4] = 1;
    state[5][5] = 1;

    for(i = 1; i <= N; i++) {
        for(j = 1; j <= N; j++) {
            event[0][T(i, j)] = state[i-1][j-1];
        }
    }

    // for(i = 0; i <= N+1; i++) {
    //     for(j = 0; j <= N+1; j++) {
    //         printf("%c", event[0][T(i, j)] + '0');
    //     }
    //     printf("\n");
    // }
    // printf("\n-----\n");

    
    runSimulations();

    // runGPUSimulations();

    for(i = 0; i <= N+1; i++) {
        for(j = 0; j <= N+1; j++) {
            if(result[T(i, j)] == 1) {
                printf("alive: x = %d, y = %d\n", i, j);
            }
        }
    }



    return 0;
}