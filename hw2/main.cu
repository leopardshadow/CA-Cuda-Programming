

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define T(x, y) ((x) + (N+2)*(y))

const int N = 5; // NxN grid
int M = 1; // M rounds

u_int8_t state[N][N] = {
    {'1','0','0','0','1'},
    {'0','0','1','0','0'},
    {'0','1','1','1','0'},
    {'0','0','1','0','0'},
    {'0','1','0','1','0'},
};

u_int8_t event[2][ (N+2) * (N+2) ] =  {0};

int simulate(int r) {
    int i, j, t;
    int nr = (r + 1) % 2;
    int adjac;
    for(i = 1; i <= N; i++) {
        for(j = 1; j <= N; j++) {
            adjac = 
                event[r][T(i-1, j-1)] + event[r][T(i, j-1)] + event[r][T(i+1, j-1)] + 
                event[r][T(i-1, j)]   +                     + event[r][T(i+1, j)]   + 
                event[r][T(i-1, j+1)] + event[r][T(i, j+1)]   + event[r][T(i+1, j+1)];


            t = event[r][T(i, j)];

    
            if((!t && adjac == 3) || (t && adjac == 2) || (t && adjac == 3)) {
                event[nr][T(i, j)] = 1;
            }
            else {
                event[nr][T(i, j)] = 0;
            }
            // printf("%d %d: %c - %d\n", i, j, '0'+event[nr][T(i, j)], adjac);
        }

    }
    return nr;
}


int runSimulations() {
    
    int r = 0;

    // do simulation m times
    for(int m = 0; m < M; m++) {

        r = simulate(r);
    }
    return r;
}

int main(int argc, char const *argv[])
{
    int i, j;
    int r;

    for(i = 1; i <= N; i++) {
        for(j = 1; j <= N; j++) {
            event[0][T(i, j)] = state[i-1][j-1] - '0';
        }
    }

    for(i = 0; i <= N+1; i++) {
        for(j = 0; j <= N+1; j++) {
            printf("%c", event[0][T(i, j)] + '0');
        }
        printf("\n");
    }
    printf("\n-----\n");

    
    r = runSimulations();

    for(i = 0; i <= N+1; i++) {
        for(j = 0; j <= N+1; j++) {
            printf("%c", event[r][T(i, j)] + '0');
        }
        printf("\n");
    }


    return 0;
}