#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

#include "parameters.h"

#define T(x, y) ((x) + (N+2)*(y))

extern float GPU_kernel(int *B, int *A);

u_int8_t state[N][N]; 
// = {
//     {'1','0','0','0','1'},
//     {'0','0','1','0','0'},
//     {'0','1','1','1','0'},
//     {'0','0','1','0','0'},
//     {'0','1','0','1','0'},
// };

u_int8_t event[2][ (N+2) * (N+2) ] =  {0};

u_int8_t *gpuEvent[2];

u_int8_t *result;

void simulate(u_int8_t *now, u_int8_t *next) {
    int i, j, t;
    int adjac;
    for(i = 1; i <= N; i++) {
        for(j = 1; j <= N; j++) {
            adjac = 
                now[T(i-1, j-1)] + now[T(i, j-1)] + now[T(i+1, j-1)] + 
                now[T(i-1, j)]   +                + now[T(i+1, j)]   + 
                now[T(i-1, j+1)] + now[T(i, j+1)] + now[T(i+1, j+1)];


            t = now[T(i, j)];

    
            if((!t && adjac == 3) || (t && adjac == 2) || (t && adjac == 3)) {
                next[T(i, j)] = 1;
            }
            else {
                next[T(i, j)] = 0;
            }
            // printf("%d %d: %c - %d\n", i, j, '0'+next[T(i, j)], adjac);
        }

    }
}


void runSimulations() {
    
    // do simulation m times
    for(int m = 0; m < M; m++) {

        simulate(event[m%2], event[(m+1)%2]);
    }
    result = event[M%2];
}

__global__ void gpu_simulate(u_int8_t *dNow, u_int8_t *dNext)
{	
    int i, j;
    int t, adjac;

    int stripeI = N / gridNum;
    int stripeJ = N / blockNum;

    for(i = 1; i <= N; i+=stripeI) {
        for(j = 1; j <= N; j+=stripeJ) {

            adjac = 
                dNow[T(i-1, j-1)] + dNow[T(i, j-1)] + dNow[T(i+1, j-1)] + 
                dNow[T(i-1, j)]   +                 + dNow[T(i+1, j)]   + 
                dNow[T(i-1, j+1)] + dNow[T(i, j+1)] + dNow[T(i+1, j+1)];
    
    
            t = dNow[T(i, j)];
        
            if((!t && adjac == 3) || (t && adjac == 2) || (t && adjac == 3)) {
                dNext[T(i, j)] = 1;
            }
            else {
                dNext[T(i, j)] = 0;
            }
        
        }
    }
    
}



void runGPUSimulations() {
    

	hipMalloc((void**)&gpuEvent[0], sizeof(u_int8_t)*((N+2)*(N+2)));
	hipMalloc((void**)&gpuEvent[1], sizeof(u_int8_t)*((N+2)*(N+2)));

    hipMemcpy(gpuEvent[0], event[0], sizeof(u_int8_t)*((N+2)*(N+2)), hipMemcpyHostToDevice);
    hipMemcpy(gpuEvent[1], event[1], sizeof(u_int8_t)*((N+2)*(N+2)), hipMemcpyHostToDevice);

    // Launch Kernel
	dim3 dimGrid(blockNum);
    dim3 dimBlock(gridNum);

    for(int m = 0; m < M; m++) {
        gpu_simulate<<<dimGrid,dimBlock>>>(gpuEvent[m%2], gpuEvent[(m+1)%2]);
    }

	hipMemcpy(event[0], gpuEvent[0], sizeof(u_int8_t)*((N+2)*(N+2)), hipMemcpyDeviceToHost);
	hipMemcpy(event[1], gpuEvent[1], sizeof(u_int8_t)*((N+2)*(N+2)), hipMemcpyDeviceToHost);

    result = event[M%2];
}



int main(int argc, char const *argv[])
{
    int i, j;

    state[150][49] = 1;
    state[150][50] = 1;
    state[150][51] = 1;

    for(i = 1; i <= N; i++) {
        for(j = 1; j <= N; j++) {
            event[0][T(i, j)] = state[i-1][j-1];
        }
    }

    // for(i = 0; i <= N+1; i++) {
    //     for(j = 0; j <= N+1; j++) {
    //         printf("%c", event[0][T(i, j)] + '0');
    //     }
    //     printf("\n");
    // }
    // printf("\n-----\n");

    
    // runSimulations();

    runGPUSimulations();

    for(i = 0; i <= N+1; i++) {
        for(j = 0; j <= N+1; j++) {
            if(result[T(i, j)] == 1) {
                printf("alive: x = %d, y = %d\n", i, j);
            }
        }
    }



    return 0;
}