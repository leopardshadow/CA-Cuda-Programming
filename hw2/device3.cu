#include "hip/hip_runtime.h"
#include "parameters.h"
#include <stdio.h>

__global__ void gpu_simulate(u_int8_t *dNow, u_int8_t *dNext)
{	
    int i, j;
    int t, adjac;

    int loopLim = N / (blockDim.x * gridDim.x);
    int head = (blockDim.x * blockIdx.x + threadIdx.x) * loopLim + 1;

    // printf("\n\nblock = %3d, thread = %3d\n", blockIdx.x, threadIdx.x);

    for(i = 1; i <= N; i++) {
        for(j = head; j < head + loopLim; j++) {

            // printf("%03d%03d\n", i, j);

            adjac = 
                dNow[T(i-1, j-1)] + dNow[T(i, j-1)] + dNow[T(i+1, j-1)] + 
                dNow[T(i-1, j)]   +                 + dNow[T(i+1, j)]   + 
                dNow[T(i-1, j+1)] + dNow[T(i, j+1)] + dNow[T(i+1, j+1)];
    
    
            t = dNow[T(i, j)];
        
            if((!t && adjac == 3) || (t && adjac == 2) || (t && adjac == 3)) {
                dNext[T(i, j)] = 1;
            }
            else {
                dNext[T(i, j)] = 0;
            }
        
        }
    }
    
}

u_int8_t *runGPUSimulations(u_int8_t event[2][(N+2)*(N+2)]) {
    
    u_int8_t *gpuEvent[2];

	hipMalloc((void**)&gpuEvent[0], sizeof(u_int8_t)*((N+2)*(N+2)));
	hipMalloc((void**)&gpuEvent[1], sizeof(u_int8_t)*((N+2)*(N+2)));

    hipMemcpy(gpuEvent[0], event[0], sizeof(u_int8_t)*((N+2)*(N+2)), hipMemcpyHostToDevice);
    hipMemcpy(gpuEvent[1], event[1], sizeof(u_int8_t)*((N+2)*(N+2)), hipMemcpyHostToDevice);

    // Launch Kernel
	dim3 dimGrid(gridNum);
    dim3 dimBlock(blockNum);

    for(int m = 0; m < M; m++) {
        gpu_simulate<<<dimGrid,dimBlock>>>(gpuEvent[m%2], gpuEvent[(m+1)%2]);
    }

	hipMemcpy(event[0], gpuEvent[0], sizeof(u_int8_t)*((N+2)*(N+2)), hipMemcpyDeviceToHost);
	hipMemcpy(event[1], gpuEvent[1], sizeof(u_int8_t)*((N+2)*(N+2)), hipMemcpyDeviceToHost);

    return event[M%2];
}
